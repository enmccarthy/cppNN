#include <hipDNN.h>
#include <cassert>
#include <cstdlib>
#include <iostream>

int convolution(){
    hipdnnHandle_t cudnn;
    checkCUDNN(hipdnnCreate(&cudnn));

    //input 
    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                        /*format=*/HIPDNN_TENSOR_NHWC,
                                        /*dataType=*/HIPDNN_DATA_FLOAT,
                                        /*batch_size=*/1,
                                        /*channels=*/3,
                                        /*image_height=*/image.rows,
                                        /*image_width=*/image.cols));

    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                                        /*format=*/HIPDNN_TENSOR_NHWC,
                                        /*dataType=*/HIPDNN_DATA_FLOAT,
                                        /*batch_size=*/1,
                                        /*channels=*/3,
                                        /*image_height=*/image.rows,
                                        /*image_width=*/image.cols));

    hipdnnFilterDescriptor_t kernel_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                                            /*dataType=*/HIPDNN_DATA_FLOAT,
                                            /*format=*/HIPDNN_TENSOR_NCHW,
                                            /*out_channels=*/3,
                                            /*in_channels=*/3,
                                            /*kernel_height=*/3,
                                            /*kernel_width=*/3));

    hipdnnConvolutionDescriptor_t convolution_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                                /*pad_height=*/1,
                                                /*pad_width=*/1,
                                                /*vertical_stride=*/1,
                                                /*horizontal_stride=*/1,
                                                /*dilation_height=*/1,
                                                /*dilation_width=*/1,
                                                /*mode=*/HIPDNN_CROSS_CORRELATION,
                                                /*computeType=*/HIPDNN_DATA_FLOAT));

    hipdnnConvolutionFwdAlgo_t convolution_algorithm;
    checkCUDNN(
        hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                            input_descriptor,
                                            kernel_descriptor,
                                            convolution_descriptor,
                                            output_descriptor,
                                            HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                            /*memoryLimitInBytes=*/0,
                                            &convolution_algorithm));

    size_t workspace_bytes = 0;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                        input_descriptor,
                                                        kernel_descriptor,
                                                        convolution_descriptor,
                                                        output_descriptor,
                                                        convolution_algorithm,
                                                        &workspace_bytes));
    std::cerr << "Workspace size: " << (workspace_bytes / 1048576.0) << "MB"
                << std::endl;
    float* h_output = new float[image_bytes];
    hipMemcpy(h_output, d_output, image_bytes, hipMemcpyDeviceToHost);
    
    // Do something with h_output ...
    
    delete[] h_output;
    hipFree(d_kernel);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_workspace);
    
    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
    
    hipdnnDestroy(cudnn);
}